#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include "gpu-new-forward.h"

#define BLOCK_SIZE 32

__global__ void conv_forward_kernel(float *y, const float *x, const float *k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    /*
    Modify this function to implement the forward pass described in Chapter 16.
    We have added an additional dimension to the tensors to support an entire mini-batch
    The goal here is to be correct AND fast.

    Function paramter definitions:
    y - output
    x - input
    k - kernel
    B - batch_size (number of images in x)
    M - number of output feature maps
    C - number of input feature maps
    H - input height dimension
    W - input width dimension
    K - kernel height and width (K x K)
    */

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;
    //(void)H_out; // silence declared but never referenced warning. remove this line when you start working
    //(void)W_out; // silence declared but never referenced warning. remove this line when you start working

    // We have some nice #defs for you below to simplify indexing. Feel free to use them, or create your own.
    // An example use of these macros:
    // float a = y4d(0,0,0,0)
    // y4d(0,0,0,0) = a

#define y4d(i3, i2, i1, i0) y[(i3) * (M * H_out * W_out) + (i2) * (H_out * W_out) + (i1) * (W_out) + i0]
#define x4d(i3, i2, i1, i0) x[(i3) * (C * H * W) + (i2) * (H * W) + (i1) * (W) + i0]
#define k4d(i3, i2, i1, i0) k[(i3) * (C * K * K) + (i2) * (K * K) + (i1) * (K) + i0]

    // Insert your GPU convolution kernel code here
    int W_grid = ceil(1.0 * W / BLOCK_SIZE);
    int m = blockIdx.x;
    int h = (blockIdx.y / W_grid) * BLOCK_SIZE + threadIdx.y;
    int w = (blockIdx.y % W_grid) * BLOCK_SIZE + threadIdx.x;
    
    int b = blockIdx.z;

    //TODO: take care of boundary conditions!

    float acc = 0.0f;

    for (int c = 0; c < C; c++) {               // Sum over all input channels/feature maps
        for (int p = 0; p < K; p++) {           // Loop over KxK filter
            for (int q = 0; q < K; q++) {
                acc += x4d(b, c, h + p, w + q) * k4d(m, c, p, q);
            }
        }
    }

    y4d(b, m, h, w) = acc;

#undef y4d
#undef x4d
#undef k4d
}

	
__host__ void GPUInterface::conv_forward_gpu_prolog(const float *host_y, const float *host_x, const float *host_k, float **device_y_ptr, float **device_x_ptr, float **device_k_ptr, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Allocate memory and copy over the relevant data structures to the GPU

    // We pass double pointers for you to initialize the relevant device pointers,
    //  which are passed to the other two functions.

    // Useful snippet for error checking
    // hipError_t error = hipGetLastError();
    // if(error != hipSuccess)
    // {
    //     std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
    //     exit(-1);
    // }
    hipError_t error;

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    int outputSize = B * M * H_out * W_out;
    int inputSize = B * C * H * W;
    int kernelSize = M * C * K * K;

    // Allocate Output Array
    hipMalloc((void **) device_y_ptr, outputSize * sizeof(float));
    error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }
    

    // Allocate Input Array
    hipMalloc((void **) device_x_ptr, inputSize * sizeof(float));
    error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }

    // Allocate Kernel Filter Array
    hipMalloc((void **) device_k_ptr, kernelSize * sizeof(float));
    error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }

    // Copy Output Array to Device
    hipMemcpy(device_y_ptr, host_y, outputSize * sizeof(float), hipMemcpyHostToDevice);
    error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }

    // Copy Input Array to Device
    hipMemcpy(device_x_ptr, host_x, inputSize * sizeof(float), hipMemcpyHostToDevice);
    error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }

    // Copy Kernel Array to Device
    hipMemcpy(device_k_ptr, host_k, kernelSize * sizeof(float), hipMemcpyHostToDevice);
    error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }
}


__host__ void GPUInterface::conv_forward_gpu(float *device_y, const float *device_x, const float *device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    // Set the kernel dimensions and call the kernel
    int W_grid = ceil(1.0 * W / BLOCK_SIZE);    // Number of horizontal tiles for output maps
    int H_grid = ceil(1.0 * H / BLOCK_SIZE);    // Numer of vertical tiles for output maps

    int Y_grid = H_grid * W_grid;   // Y-Dimension of the grid

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE, 1);
    dim3 gridDim(M, Y_grid, B);

    conv_forward_kernel<<<gridDim, blockDim>>>(device_y, device_x, device_k, B, M, C, H, W, K);
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }
}


__host__ void GPUInterface::conv_forward_gpu_epilog(float *host_y, float *device_y, float *device_x, float *device_k, const int B, const int M, const int C, const int H, const int W, const int K)
{
    hipError_t error;

    const int H_out = H - K + 1;
    const int W_out = W - K + 1;

    int outputSize = B * M * H_out * W_out;

    // Copy the output back to host
    hipMemcpy(host_y, device_y, outputSize * sizeof(float), hipMemcpyDeviceToHost);
    error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }

    // Free device memory
    hipFree(device_y);
    error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }

    hipFree(device_x);
    error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }

    hipFree(device_k);
    error = hipGetLastError();
    if(error != hipSuccess)
    {
        std::cout<<"CUDA error: "<<hipGetErrorString(error)<<std::endl;
        exit(-1);
    }
}


__host__ void GPUInterface::get_device_properties()
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for(int dev = 0; dev < deviceCount; dev++)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, dev);

        std::cout<<"Device "<<dev<<" name: "<<deviceProp.name<<std::endl;
        std::cout<<"Computational capabilities: "<<deviceProp.major<<"."<<deviceProp.minor<<std::endl;
        std::cout<<"Max Global memory size: "<<deviceProp.totalGlobalMem<<std::endl;
        std::cout<<"Max Constant memory size: "<<deviceProp.totalConstMem<<std::endl;
        std::cout<<"Max Shared memory size per block: "<<deviceProp.sharedMemPerBlock<<std::endl;
        std::cout<<"Max threads per block: "<<deviceProp.maxThreadsPerBlock<<std::endl;
        std::cout<<"Max block dimensions: "<<deviceProp.maxThreadsDim[0]<<" x, "<<deviceProp.maxThreadsDim[1]<<" y, "<<deviceProp.maxThreadsDim[2]<<" z"<<std::endl;
        std::cout<<"Max grid dimensions: "<<deviceProp.maxGridSize[0]<<" x, "<<deviceProp.maxGridSize[1]<<" y, "<<deviceProp.maxGridSize[2]<<" z"<<std::endl;
        std::cout<<"Warp Size: "<<deviceProp.warpSize<<std::endl;
    }
}
